#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <sys/time.h>		// get time of day
#include <sys/times.h>		// get time of day
#include <sys/mman.h>		// mmap
#include <unistd.h>		// getpid
#include <hip/hip_runtime.h>
// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define PRINT_TIME 				1
#define SM_ARR_LEN			2048
#define TOL						1e-1
#define OMEGA 1.8
#define GET_SECONDS_TICS 100	
#define IMUL(a, b) __mul24(a, b)
#define NUM_THREADS_PER_BLOCK_DIMENSION 	16
#define NUM_BLOCKS_PER_GRID_DIMENSION SM_ARR_LEN/NUM_THREADS_PER_BLOCK_DIMENSION   // ensures one output per thread

void initializeArray1D(float *arr, int len, int seed); // Initialize 2D array as concatenated sets of 1D arrays
void mmm_kij(float* a0, float* b0, float* c0);

double get_seconds() { 	/* routine to read time */
    struct tms rusage;
    times(&rusage);	/* UNIX utility: time in clock ticks */
    return (double)(rusage.tms_utime)/(double)(GET_SECONDS_TICS);
}


__global__ void kernel_MMM (int arrLen, float* A, float* B, float* C) {
int idx = blockDim.x*blockIdx.x + threadIdx.x;
int idy = blockDim.y*blockIdx.y + threadIdx.y;
      int index = idx*arrLen+idy;
      if (index < arrLen*arrLen){
        C[index] = 0.0;
        float temp = 0.0;
        for (int k = 0; k < arrLen; k++)
          temp += (A[idx*arrLen+k] * B[k*arrLen+idy]);
        C[index] = temp;
      }
}

int main(int argc, char **argv){
	int arrLen = 0;

	// GPU Timing variables
	hipEvent_t start, stop;
	float elapsed_gpu;
	double sec;
	// Arrays on GPU global memoryc
	float *d_arrayA;
	float *d_arrayB;
	float *d_arrayC;

	// Arrays on the host memory
	float *h_arrayA;
	float *h_arrayB;
	float *h_arrayC_CPU;
	float *h_arrayC_GPU;
 
	int i, errCount = 0, zeroCount = 0;
	
	if (argc > 1) {
		arrLen  = atoi(argv[1]);
	}
	else {
		arrLen = SM_ARR_LEN;
	}

	printf("Length of the array = %d\n", arrLen);

  // Select GPU
  CUDA_SAFE_CALL(hipSetDevice(0));
  
  // Set block dimensions
  dim3 threadsPerBlock(NUM_THREADS_PER_BLOCK_DIMENSION, NUM_THREADS_PER_BLOCK_DIMENSION);
  dim3 NUM_BLOCKS(NUM_BLOCKS_PER_GRID_DIMENSION,NUM_BLOCKS_PER_GRID_DIMENSION);


	// Allocate GPU memory
	size_t allocSize = arrLen * arrLen * sizeof(float);
	CUDA_SAFE_CALL(hipMalloc((void **)&d_arrayA, allocSize));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_arrayB, allocSize));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_arrayC, allocSize));
		
	// Allocate arrays on host memory
	h_arrayA                        = (float *) malloc(allocSize);
	h_arrayB                        = (float *) malloc(allocSize);
	h_arrayC_CPU                        = (float *) malloc(allocSize);
	h_arrayC_GPU                        = (float *) malloc(allocSize);
	
 
	// Initialize the host arrays
	printf("\nInitializing the arrays ...");
	// Arrays are initialized with a known seed for reproducability
	initializeArray1D(h_arrayA, arrLen, 2453);
	initializeArray1D(h_arrayB, arrLen, 2453);
	printf("\t... done\n\n");
	
  #if PRINT_TIME
	// Create the cuda events
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif

	// Transfer the arrays to the GPU memory
	CUDA_SAFE_CALL(hipMemcpy(d_arrayA, h_arrayA, allocSize, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_arrayB, h_arrayB, allocSize, hipMemcpyHostToDevice));
	

	// Launch the kernel
	kernel_MMM <<<NUM_BLOCKS, threadsPerBlock >>>(arrLen, d_arrayA, d_arrayB, d_arrayC);
  
	// Check for errors during launch
	CUDA_SAFE_CALL(hipPeekAtLastError());
   	
	// Transfer the results back to the host
	CUDA_SAFE_CALL(hipMemcpy(h_arrayC_GPU, d_arrayC, allocSize, hipMemcpyDeviceToHost));
 
 
#if PRINT_TIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_gpu, start, stop);
	printf("\nGPU time: %f (msec)\n", elapsed_gpu);
	hipEventDestroy(start);
	hipEventDestroy(stop);
#endif
	
	// Compute the results on the host

 
  printf("Starting CPU Computation\n");
 
  sec = get_seconds();		
  mmm_kij(h_arrayA, h_arrayB, h_arrayC_CPU);
  sec = (get_seconds() - sec);	
  printf("\n CPUTime = %f (msec)\n", sec*1000.0);

   
   
  
	printf("Comparing Results\n");
	// Compare the results
	for(i = 0; i < arrLen*arrLen; i++) {
		if (abs((h_arrayC_CPU[i] - h_arrayC_GPU[i])/h_arrayC_GPU[i]) > TOL) {
			errCount++;
		}
		if (h_arrayC_GPU[i] == 0) {
			zeroCount++;
		}
	}
	
	
	if (errCount > 0) {
		printf("\n@ERROR: TEST FAILED: %d results did not matched\n", errCount);
	}
	else if (zeroCount > 0){
		printf("\n@ERROR: TEST FAILED: %d results (from GPU) are zero\n", zeroCount);
	}
	else {
		printf("\nTEST PASSED: All results matched\n");
	}
 
	
	// Free-up device and host memory
	CUDA_SAFE_CALL(hipFree(d_arrayA));
	CUDA_SAFE_CALL(hipFree(d_arrayB));
	CUDA_SAFE_CALL(hipFree(d_arrayC));
		   
	free(h_arrayC_CPU);
	free(h_arrayC_GPU);
	free(h_arrayA);
	free(h_arrayB);
		
	return 0;
}

void initializeArray1D(float *arr, int len, int seed) {
	int i;
	float randNum;
	srand(seed);

	for (i = 0; i < len*len; i++) {
		randNum = (float) rand();
		arr[i] = randNum;
   
	}
}


void mmm_kij(float* a0, float* b0, float* c0)
{
  long int i, j, k;
  float r;
  int length = SM_ARR_LEN;
  for (k = 0; k < length; k++)
    for (i = 0; i < length; i++) {
      r = a0[i*length+k];
      for (j = 0; j < length; j++)
	c0[i*length+j] += r*b0[k*length+j];
    }
}